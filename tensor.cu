#include "hip/hip_runtime.h"
// Includes

#include <luaT.h>
#include <TH.h>

#include <THC/THC.h>

#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/inner_product.h>
#include <thrust/device_ptr.h>

#include "tensor.h"

#include <sstream>





#define cudaAssert(ans) { cudaAssert_((ans), __FILE__, __LINE__); }
inline void cudaAssert_(hipError_t code, const char *file, int line)
{

  if (code != hipSuccess) 
  {
    std::ostringstream out;
    out << "cuda error " << file << ":" << line << " " << hipGetErrorString(code);
    
    throw std::logic_error(out.str());
    
  }
}

template<typename Op>
__global__ void kernel_indexReduce(
   float *res, float *src, long* res_stride, float *index,
   long res_nDim, int dim, long idx_size, long src_size, long size_dim, Op const &op
)
{
  int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

  long flat_size = src_size / idx_size;

  if (thread_idx < flat_size)
  {
    long coeff = 0;
    for (int i=0; i<idx_size; i++)
    {
      int leftover = thread_idx;
      int targetIdx = 0;
      int resIdx = 0;
      for (int d=0; d<res_nDim; d++)
      {
        if (d < dim)
        {
          long stride_d = res_stride[d] / size_dim;
          coeff = leftover / stride_d;
          leftover -= coeff * stride_d;
          targetIdx += coeff * stride_d * idx_size;
          resIdx += coeff * res_stride[d];
        }
        else if (d > dim)
        {
          coeff = leftover / res_stride[d];
          leftover -= coeff * res_stride[d];
          targetIdx += coeff * res_stride[d];
          resIdx += coeff * res_stride[d];
        }
      }
      
      int r = resIdx + ((int)(index[i])-1)*res_stride[dim];
      op(res[r], src[targetIdx + i*res_stride[dim]]);
    }
  }
}

template<typename Op>
void indexReduce(THCState *state, THCudaTensor *res_, int dim, THLongTensor *indices, THCudaTensor *src, Op const &op)
{
  THCudaTensor *indices_;
  long *stride_;
  long nIndex = indices->size[0];
  long nSrc;

  THArgCheck(indices->nDimension == 1, 3, "expecting vector of indices");
  THArgCheck(dim < src->nDimension, 4, "Indexing dim is out of bounds");
  THArgCheck(src->nDimension > 0, 2, "Source tensor is empty");
  THArgCheck(nIndex == src->size[dim], 4, "length of src.size[dim] is not equal to length of indices");

  src = THCudaTensor_newContiguous(state, src);
  indices_ = THCudaTensor_newWithSize1d(state, nIndex);
  THCudaTensor_copyLong(state, indices_, indices);

  nSrc = THCudaTensor_nElement(state, src);
  
  const int size = 16;
  dim3 nthreads(size, size);
  dim3 nblocks(ceil((float)(nSrc / nIndex) / (size*size)));
  

  THCudaCheck(hipMalloc((void**)&stride_, res_->nDimension * sizeof(long)));
  THCudaCheck(hipMemcpy(stride_, res_->stride, res_->nDimension * sizeof(long), hipMemcpyHostToDevice));

  kernel_indexReduce<<<nblocks, nthreads>>>(
    THCudaTensor_data(state, res_), THCudaTensor_data(state, src),
    stride_, THCudaTensor_data(state, indices_),
    res_->nDimension, dim, nIndex,
    THCudaTensor_nElement(state, src), res_->size[dim], op
  );

  THCudaCheck(hipFree(stride_));
  THCudaTensor_free(state, indices_);
  THCudaTensor_free(state, src);
}


template<typename Op>
void transform(THCState * state, THCudaTensor *self_, THCudaTensor *src_, Op const &op)
{
  THCudaTensor_resizeAs(state, self_, src_);
  THCudaTensor *self = THCudaTensor_newContiguous(state, self_);
  THCudaTensor *src = THCudaTensor_newContiguous(state, src_);
  long size = THCudaTensor_nElement(state, self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(state, self));
  thrust::device_ptr<float> src_data(THCudaTensor_data(state, src));

  thrust::transform(src_data, src_data+size, self_data, op);

  THCudaTensor_free(state, src);
  THCudaTensor_freeCopyTo(state, self, self_);
}



struct add_functor {
   
  __device__ void operator()(float& x, float& y) const {
    x += y;
  }  
};



void libtensor_Cuda_clamp(THCState *state, THCudaTensor *self_, THCudaTensor *src_, float lower, float upper) {
  transform(state, self_, src_, clamp_functor<float>(lower, upper));
}


void libtensor_Cuda_min(THCState *state, THCudaTensor *self_, THCudaTensor *src_, float lower) {
  transform(state, self_, src_, min_functor<float>(lower));
}

void libtensor_Cuda_max(THCState *state, THCudaTensor *self_, THCudaTensor *src_, float upper) {
  transform(state, self_, src_, max_functor<float>(upper));
}


void libtensor_Cuda_mod(THCState *state, THCudaTensor *self_, THCudaTensor *src_, float p) {
  transform(state, self_, src_, mod_functor<float>(p));
}


void libtensor_Cuda_indexSum(THCState *state, THCudaTensor *res_, int dim, THLongTensor *indices, THCudaTensor *src) {
  indexReduce(state, res_, dim, indices, src, add_functor());
}


